#include "hip/hip_runtime.h"
// clang-format off
#include <array>
#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <chrono>
/*
#undef __HIPCC__
#undef __CUDACC_VER__
#undef __CUDA_ARCH__
#include <Eigen/Eigen>
*/
#include <hip/hip_runtime.h>
#include <utils/hip/hip_runtime_api.h>
// clang-format on

namespace {
template <typename T> T *create2DArrayCPU(size_t n_rows, size_t n_cols) {
  T *ptr = new T[n_rows * n_cols];
  return ptr;
}

template <typename T> T *create2DArrayGPU(size_t n_rows, size_t n_cols) {
  T *ptr = nullptr;
  checkCudaErrors(hipMalloc(ptr, sizeof(T) * n_rows * n_cols));
  return ptr;
}

} // namespace

template <typename T> class MatCPU {
public:
  MatCPU(size_t height, size_t width)
      : data_(height * width, T(0)), height_(height), width_(width) {}
  MatCPU(const std::vector<T> &data, size_t height, size_t width)
      : data_(data), height_(height), width_(width) {}
  MatCPU(std::vector<T> &&data, size_t height, size_t width)
      : data_(data), height_(height), width_(width) {}

  T &at(size_t y, size_t x) { return data_[y * width() + x]; }
  const T &at(size_t y, size_t x) const { return data_[y * width() + x]; }

  static MatCPU<float> randn(size_t n_rows, size_t n_cols) {
    size_t n = n_rows * n_cols;
    std::vector<float> data(n);
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0.0, 1.0);
    for (size_t i = 0; i < n; i += 1) {
      data[i] = distribution(generator);
    }
    return MatCPU<float>(std::move(data), n_rows, n_cols);
  }

  friend MatCPU<T> operator*(const MatCPU<T> &A, const MatCPU<T> &B) {
    MatCPU<T> C(A.height(), B.width());
    for (size_t y = 0; y < C.height(); y += 1) {
      for (size_t x = 0; x < C.width(); x += 1) {
        assert(A.width() == B.height());
        for (size_t i = 0; i < A.width(); i += 1) {
          C.at(y, x) += A.at(y, i) * B.at(i, x);
        }
      }
    }
    return C;
  }

  size_t height() const { return height_; }
  size_t width() const { return width_; }

private:
  std::vector<T> data_;
  size_t height_, width_;
};

/*
void test_cpu_matmul() {
  MatCPU<float> A = MatCPU<float>::randn(1024, 512);
  MatCPU<float> B = MatCPU<float>::randn(512, 256);
  MatCPU<float> C = A * B;
  //
  Eigen::MatrixXf eA(1024, 512);
  for (size_t y = 0; y < A.height(); y += 1) {
    for (size_t x = 0; x < A.width(); x += 1) {
      eA(y, x) = A.at(y, x);
    }
  }
  Eigen::MatrixXf eB(512, 1024);
  for (size_t y = 0; y < B.height(); y += 1) {
    for (size_t x = 0; x < B.width(); x += 1) {
      eB(y, x) = B.at(y, x);
    }
  }
  Eigen::MatrixXf eC = eA * eB;
  //
  for (size_t y = 0; y < C.height(); y += 1) {
    for (size_t x = 0; x < C.width(); x += 1) {
      assert(eC(y, x) == C.at(y, x));
    }
  }
  std::cout << "[PASS] test_matmul()" << std::endl;
}
*/

namespace {
#define ERROR_ALLOW 1e-3
#define BLOCK_SIZE 32
#define GRID_SIZE 64
const dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
const dim3 blocksPerGrid(GRID_SIZE, GRID_SIZE, 1);
#define HANDLE_ERROR(x) checkCudaErrors(x)

/*********************************************/
/***************without cache*****************/
/*********************************************/
__global__ void matrixMul_kernel_1(const float *A, const float *B, float *C,
                                   uint32_t hA, uint32_t wA, uint32_t hB,
                                   uint32_t wB) {
  for (uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; y < hA;
       y += gridDim.y * blockDim.y) {
    for (uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; x < wB;
         x += gridDim.x * blockDim.x) {
      //
      uint32_t iC = y * wB + x;
      C[iC] = 0.0f;

      //
      for (uint32_t offset = 0; offset < wA; offset++) {
        uint32_t iA = y * wA + offset;
        uint32_t iB = offset * wB + x;
        C[iC] += A[iA] * B[iB];
      }
    }
  }
}

void matrixMul_gpu_1(const float *A, const float *B, float *C, uint32_t hA,
                     uint32_t wA, uint32_t hB, uint32_t wB) {
  uint32_t sizeA = hA * wA * sizeof(float);
  uint32_t sizeB = hB * wB * sizeof(float);
  uint32_t sizeC = hA * wB * sizeof(float);

  // allocate device memory
  float *dev_A = NULL, *dev_B = NULL, *dev_C = NULL;
  HANDLE_ERROR(hipMalloc(&dev_A, sizeA));
  HANDLE_ERROR(hipMalloc(&dev_B, sizeB));
  HANDLE_ERROR(hipMalloc(&dev_C, sizeC));

  // copy from host to device
  HANDLE_ERROR(hipMemcpy(dev_A, A, sizeA, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_B, B, sizeB, hipMemcpyHostToDevice));

  // real timer
  clock_t before = clock();
  matrixMul_kernel_1<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,
                                                         hA, wA, hB, wB);
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());
  printf("gpu duration: %ld\n", clock() - before);

  // copy from device to host
  HANDLE_ERROR(hipMemcpy(C, dev_C, sizeC, hipMemcpyDeviceToHost));

  // free memory
  HANDLE_ERROR(hipFree(dev_A));
  HANDLE_ERROR(hipFree(dev_B));
  HANDLE_ERROR(hipFree(dev_C));
}

/*********************************************/
/*****************with cache******************/
/*********************************************/
__global__ void matrixMul_kernel_2(const float *A, const float *B, float *C,
                                   uint32_t hA, uint32_t wA, uint32_t hB,
                                   uint32_t wB) {
  for (uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; y < hA;
       y += gridDim.y * blockDim.y) {
    for (uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; x < wB;
         x += gridDim.x * blockDim.x) {

      //
      float value = 0.0f;
      for (uint32_t offset = 0; offset < wA; offset++) {
        uint32_t iA = y * wA + offset;
        uint32_t iB = offset * wB + x;
        value += A[iA] * B[iB];
      }
      uint32_t iC = y * wB + x;
      C[iC] = value;
    }
  }
}

void matrixMul_gpu_2(const float *A, const float *B, float *C, uint32_t hA,
                     uint32_t wA, uint32_t hB, uint32_t wB) {
  uint32_t sizeA = hA * wA * sizeof(float);
  uint32_t sizeB = hB * wB * sizeof(float);
  uint32_t sizeC = hA * wB * sizeof(float);

  // allocate device memory
  float *dev_A = NULL, *dev_B = NULL, *dev_C = NULL;
  HANDLE_ERROR(hipMalloc(&dev_A, sizeA));
  HANDLE_ERROR(hipMalloc(&dev_B, sizeB));
  HANDLE_ERROR(hipMalloc(&dev_C, sizeC));

  // copy from host to device
  HANDLE_ERROR(hipMemcpy(dev_A, A, sizeA, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_B, B, sizeB, hipMemcpyHostToDevice));

  // real JIT
  clock_t before = clock();
  matrixMul_kernel_2<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,
                                                         hA, wA, hB, wB);
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());
  printf("gpu duration: %ld\n", clock() - before);

  // copy from device to host
  HANDLE_ERROR(hipMemcpy(C, dev_C, sizeC, hipMemcpyDeviceToHost));

  // free memory
  HANDLE_ERROR(hipFree(dev_A));
  HANDLE_ERROR(hipFree(dev_B));
  HANDLE_ERROR(hipFree(dev_C));
}

/*********************************************/
/****************with shared******************/
/*********************************************/
__global__ void matrixMul_kernel_3(const float *A, const float *B, float *C,
                                   uint32_t hA, uint32_t wA, uint32_t hB,
                                   uint32_t wB) {
  // slide window on C
  for (uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y; gy < hA;
       gy += gridDim.y * blockDim.y) {
    for (uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x; gx < wB;
         gx += gridDim.x * blockDim.x) {
      float value = 0.0f;
      // slide window on A and B
      for (uint32_t offset = 0; offset < wA; offset += BLOCK_SIZE) {
        __shared__ float sA[BLOCK_SIZE * BLOCK_SIZE];
        __shared__ float sB[BLOCK_SIZE * BLOCK_SIZE];
        sA[threadIdx.y * BLOCK_SIZE + threadIdx.x] =
            A[gy * wA + (offset + threadIdx.x)];
        sB[threadIdx.y * BLOCK_SIZE + threadIdx.x] =
            B[(offset + threadIdx.y) * wB + gx];
        __syncthreads();
        for (uint32_t i = 0; i < BLOCK_SIZE; i += 1) {
          value += sA[threadIdx.y * BLOCK_SIZE + i] *
                   sB[i * BLOCK_SIZE + threadIdx.x];
        }
        __syncthreads();
      }
      C[gy * wB + gx] = value;
    }
  }
}

void matrixMul_gpu_3(const float *A, const float *B, float *C, uint32_t hA,
                     uint32_t wA, uint32_t hB, uint32_t wB) {
  uint32_t sizeA = hA * wA * sizeof(float);
  uint32_t sizeB = hB * wB * sizeof(float);
  uint32_t sizeC = hA * wB * sizeof(float);

  // allocate device memory
  float *dev_A = NULL, *dev_B = NULL, *dev_C = NULL;
  HANDLE_ERROR(hipMalloc(&dev_A, sizeA));
  HANDLE_ERROR(hipMalloc(&dev_B, sizeB));
  HANDLE_ERROR(hipMalloc(&dev_C, sizeC));

  // copy from host to device
  HANDLE_ERROR(hipMemcpy(dev_A, A, sizeA, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_B, B, sizeB, hipMemcpyHostToDevice));

  // real JIT
  clock_t before = clock();
  matrixMul_kernel_3<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,
                                                         hA, wA, hB, wB);
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());
  printf("gpu duration: %ld\n", clock() - before);

  // copy from device to host
  HANDLE_ERROR(hipMemcpy(C, dev_C, sizeC, hipMemcpyDeviceToHost));

  // free memory
  HANDLE_ERROR(hipFree(dev_A));
  HANDLE_ERROR(hipFree(dev_B));
  HANDLE_ERROR(hipFree(dev_C));
}

__global__ void matrixMul_kernel_4(const float *A, const float *B, float *C,
                                   uint32_t hA, uint32_t wA, uint32_t hB,
                                   uint32_t wB) {
  // slide window on C
  for (uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y; gy < hA;
       gy += gridDim.y * blockDim.y) {
    for (uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x; gx < wB;
         gx += gridDim.x * blockDim.x) {
      // slide window on A and B
      for (uint32_t offset = 0; offset < wA; offset += BLOCK_SIZE) {
        __shared__ float sA[BLOCK_SIZE * BLOCK_SIZE];
        __shared__ float sB[BLOCK_SIZE * BLOCK_SIZE];
        sA[threadIdx.y * BLOCK_SIZE + threadIdx.x] =
            A[gy * wA + (offset + threadIdx.x)];
        sB[threadIdx.y * BLOCK_SIZE + threadIdx.x] =
            B[(offset + threadIdx.y) * wB + gx];
        __syncthreads();
        for (uint32_t i = 0; i < BLOCK_SIZE; i += 1) {
          C[gy * wB + gx] += sA[threadIdx.y * BLOCK_SIZE + i] *
                             sB[i * BLOCK_SIZE + threadIdx.x];
        }
        __syncthreads();
      }
    }
  }
}

void matrixMul_gpu_4(const float *A, const float *B, float *C, uint32_t hA,
                     uint32_t wA, uint32_t hB, uint32_t wB) {
  uint32_t sizeA = hA * wA * sizeof(float);
  uint32_t sizeB = hB * wB * sizeof(float);
  uint32_t sizeC = hA * wB * sizeof(float);

  // allocate device memory
  float *dev_A = NULL, *dev_B = NULL, *dev_C = NULL;
  HANDLE_ERROR(hipMalloc(&dev_A, sizeA));
  HANDLE_ERROR(hipMalloc(&dev_B, sizeB));
  HANDLE_ERROR(hipMalloc(&dev_C, sizeC));

  // copy from host to device
  HANDLE_ERROR(hipMemcpy(dev_A, A, sizeA, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_B, B, sizeB, hipMemcpyHostToDevice));

  // real JIT
  clock_t before = clock();
  matrixMul_kernel_4<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,
                                                         hA, wA, hB, wB);
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());
  printf("gpu duration: %ld\n", clock() - before);

  // copy from device to host
  HANDLE_ERROR(hipMemcpy(C, dev_C, sizeC, hipMemcpyDeviceToHost));

  // free memory
  HANDLE_ERROR(hipFree(dev_A));
  HANDLE_ERROR(hipFree(dev_B));
  HANDLE_ERROR(hipFree(dev_C));
}

/*********************************************/
/*********************cpu*********************/
/*********************************************/
void matrixMul_cpu(const float *A, const float *B, float *C, size_t hA,
                   size_t wA, size_t hB, size_t wB) {
  if (wA != hB) {
    fprintf(stderr, "matrix error, wA: %ld, hB: %ld\n", wA, hB);
    return;
  }

  clock_t before = clock();
  for (size_t rA = 0; rA < hA; rA++) {
    for (size_t cB = 0; cB < wB; cB++) {
      //
      size_t iC = (rA * wB) + cB;
      C[iC] = 0.0f;

      //
      for (size_t offset = 0; offset < wA; offset++) {
        size_t iA = rA * wA + offset;
        size_t iB = (offset * wB) + cB;

        C[iC] += A[iA] * B[iB];
      }
    }
  }
  printf("cpu duration: %ld\n", clock() - before);
}

float *randMatrix(size_t height, size_t width) {
  size_t size = height * width * sizeof(float);
  float *dst = (float *)malloc(size);

  for (size_t i = 0; i < height * width; i++) {
    dst[i] = (float)rand() / RAND_MAX;
  }

  return dst;
}

bool allClose(float *A, float *B, size_t height, size_t width) {
  for (size_t row = 0; row < height; row++) {
    for (size_t col = 0; col < width; col++) {
      size_t idx = row * width + col;
      if (!isfinite(A[idx]) or !isfinite(B[idx])) {
        fprintf(stderr, "infinite error, idx: %ld\n", idx);
        return false;
      }
      if (fabs(A[idx] - B[idx]) > ERROR_ALLOW) {
        fprintf(stderr, "inequal error, %f != %f, (x, y): (%ld, %ld)\n", A[idx],
                B[idx], col, row);
        return false;
      }
    }
  }
  return true;
}

}; // namespace

void do_benchmark() {
  size_t hA = 8192, wA = 8192, hB = 8192, wB = 8192;

  float *A = randMatrix(hA, wA);
  float *B = randMatrix(hB, wB);
  float *C1 = randMatrix(hA, wB), *C2 = randMatrix(hA, wB),
        *C3 = randMatrix(hA, wB), *C4 = randMatrix(hA, wB),
        *C5 = randMatrix(hA, wB);

  // matrixMul_cpu(A, B, C1, hA, wA, hB, wB);
  matrixMul_gpu_1(A, B, C2, hA, wA, hB, wB);
  matrixMul_gpu_2(A, B, C3, hA, wA, hB, wB);
  matrixMul_gpu_3(A, B, C4, hA, wA, hB, wB);
  matrixMul_gpu_4(A, B, C5, hA, wA, hB, wB);
  // allClose(C1, C2, hA, wB);
  // allClose(C2, C3, hA, wB);
  // allClose(C3, C4, hA, wB);

  free(A);
  free(B);
  free(C1);
  free(C2);
  free(C3);
  free(C4);
  free(C5);

  printf("all down\n");
}

int main() {
  for (size_t i = 0; i < 16; i += 1) {
    do_benchmark();
  }
  return 0;
}

/*
int main() {
  test_cpu_matmul();
  return 0;
}
*/